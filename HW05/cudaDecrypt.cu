#include "hip/hip_runtime.h"
include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "functions.h"


int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;
  unsigned int Nchars;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

	// Read in cyphertexts.
	FILE *cypher; // Declare a pointer for messages.txt.
	cypher = fopen("message.txt", "r"); // Read messages.txt

	// Handle file not being found.
	if (cypher == NULL)
	{
		printf("File not found. Aborting program. \n");
		exit(1);
	}//end if
	fscanf(cypher, "%u", &Nints); // Grab number of cyphertexts.

	// Read in public key data.
	FILE *key;
	key = fopen("public_key.txt", "r");

	// File not found handling.
	if (key == NULL)
	{
		printf("File not found. Aborting program. \n");
		exit(1);
	}// end if
	fscanf(key, "%u", &n); // Read in n.
	fscanf(key, "%u", &p); // Read in p.
	fscanf(key, "%u", &g); // Read in g.
	fscanf(key, "%u", &h); // Read in h.
	fclose(key); // Close public_key.txt

	// Allocate memory for cyphertexts.
	unsigned int *m = malloc(Nints*sizeof(unsigned int));
	unsigned int *a = malloc(Nints*sizeof(unsigned int));

	// Scan in cypher texts.
	for (int i = 0; i < Nints; i++)
	{
		fscanf(cypher, "%u", &m[i]);
		fscanf(cypher, "%u", &a[i]);
	}//end for
	fclose(cypher); // Close messages.txt

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
	printf("%g total values tested. \n", work);
  }

  /* Q3 After finding the secret key, decrypt the message */
  unsigned int buffer = 1024;
  unsigned int cpi = (n-1)/8;
  Nchars = Nints*cpi;
  unsigned char *message = malloc(Nchars*sizeof(unsigned char)); // Allocate space for string
  ElGamalDecrypt(m, a, Nints, p, x); // Decrypt message.
  convertZToString(m, Nints, message, Nchars); // Convert decrypted message to a string.
	
  // Print the message to the terminal.
  printf("Decrypted message: \"%s\"\n ", message);
  /*
  for (int i = 0; i <= Nints; i++)
  {
  	printf("%c", message[i]);
  }//end for
  printf("\n");
  */
  free(message);
  return 0;
}
