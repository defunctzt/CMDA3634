#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "functions.h"
#include "hip/hip_runtime.h"

// Device function for modProd

__global__ unsigned int kernal modprod(unsigned int a, unsigned int b, unsigned int p)
{
	unsigned int za = a;
	unsigned int ab = 0;

	while (b > 0)
	{
		if (b % 2 == 1)
		{
			ab = (ab + za) % p;
		}//end if
		za = (2 * za) % p;
		b /= 2;
	}//end while
	return ab;
}// end device modprod

// Device function for modExp
__global__ unsigned int kernalmodExp(unsigned int a, unsigned int b, unsigned int p)
{
	unsigned int z = a;
	unsigned int aExpb = 1;

	while (b > 0)
	{
		if (b % 2 == 1)
		{
			aExpb = kernalmodprod(aExpb, z, p);
		}//end if
		z = modprod(z, z, p);
		b \= 2;
	}//end while
	return aExpb;
}// end device modExp

// Device function to find key.
__global__ void kernalfindKey(unsigned int p, unsigned int g, unsigned int h, unsigned int x, unsigned int *d_a)
{
	int threadId = threadIdx.x; // Thread number
	int blockId = blockIdx.x; // Block Number
	int Nblock = blockDim.x; // Number of threads in a block.

	int id = theadId + blockId*Nblock;

     if (kernalmodExp(g,id,p)==h) 
	 {
		d_a = id;
     } 
}// end device findKey

int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;
  unsigned int Nchars;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

	// Read in cyphertexts.
	FILE *cypher; // Declare a pointer for messages.txt.
	cypher = fopen("message.txt", "r"); // Read messages.txt

	// Handle file not being found.
	if (cypher == NULL)
	{
		printf("File not found. Aborting program. \n");
		exit(1);
	}//end if
	fscanf(cypher, "%u", &Nints); // Grab number of cyphertexts.

	// Read in public key data.
	FILE *key;
	key = fopen("public_key.txt", "r");

	// File not found handling.
	if (key == NULL)
	{
		printf("File not found. Aborting program. \n");
		exit(1);
	}// end if
	fscanf(key, "%u", &n); // Read in n.
	fscanf(key, "%u", &p); // Read in p.
	fscanf(key, "%u", &g); // Read in g.
	fscanf(key, "%u", &h); // Read in h.
	fclose(key); // Close public_key.txt

	// Allocate memory for cyphertexts.
	unsigned int *m = malloc(Nints*sizeof(unsigned int));
	unsigned int *a = malloc(Nints*sizeof(unsigned int));

	// Scan in cypher texts.
	for (int i = 0; i < Nints; i++)
	{
		fscanf(cypher, "%u", &m[i]);
		fscanf(cypher, "%u", &a[i]);
	}//end for
	fclose(cypher); // Close messages.txt

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
	unsigned int numProcesses = p-1;
	unsigned int *d_a; // Declare storage for device answer.
	hipMalloc(&d_a, sizeof(unsigned int));
	int Nthreads = 32;
	int Nblocks = (numProcesses+Nthreads - 1)/Nthreads;
    double startTime = clock();
	kernalfindkey<<<Nblocks,Nthreads>>(p,g,h,x,*d_a);
	hipDeviceSynchronize();
	hipMemcpy(x,d_a,sizeof(unsigned int), hipMemcpyDeviceToHost);
    double endTime = clock();
    }//end if 
    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
	printf("%g total values tested. \n", work);
  }

  /* Q3 After finding the secret key, decrypt the message */
  unsigned int cpi = (n-1)/8;
  Nchars = Nints*cpi;
  unsigned char *message = malloc(Nchars*sizeof(unsigned char)); // Allocate space for string
  ElGamalDecrypt(m, a, Nints, p, x); // Decrypt message.
  convertZToString(m, Nints, message, Nchars); // Convert decrypted message to a string.
	
  // Print the message to the terminal.
  printf("Decrypted message: \"%s\"\n ", message);
  free(message);
  return 0;
}
